#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <cmath>
#include "a3.hpp"

/*
    Assuming that the host has atleast 1 gpu. 
    Return the maximum number of threads in a block of device 0
*/
int get_block_size() {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    return deviceProp.maxThreadsPerBlock;
}

__global__ void gaussian_kde_kernel(int n, float h, const float *x, float *y) {
    extern __shared__ float sdata[];

    int gidx = blockIdx.x * blockDim.x + threadIdx.x;
    int lidx = threadIdx.x;
    float sum = 0.0;

    for (int i = 0; i < gridDim.x; i++) {
        // save blockwise data into the shared memory in each iteration
        if ((lidx + i * blockDim.x) < n )
            sdata[lidx] = x[(lidx + i * blockDim.x)];

        // sync threads before computation
        __syncthreads();

        /*
            To compute y[k], take x[k] from global memory. 
            Calculate the cummulative sum of elements till the block reached in this iteration
        */
        if (gidx < n) {
            int j = 0;
            while ((j < blockDim.x) && ((j + i * blockDim.x) < n)) {
                float diff = (x[gidx] - sdata[j]) / h;
                sum += (expf(-0.5 * diff * diff) / (sqrtf(2.0 * M_PI)));
                j++;
            }
            
        }

        // Sync threads before overriding values of next block into shared memory.
        __syncthreads();
    }

    y[gidx] = sum / (n * h);

}

void gaussian_kde(int n, float h, const std::vector<float>& x, std::vector<float>& y) {

    // Determine block and grid size
    int blockSize = get_block_size();
    int numBlocks = ( n + blockSize - 1 ) / blockSize;

    // Allocate and Load Memory
    float *d_x, *d_y;
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    hipMemcpy(d_x, x.data(), n * sizeof(float), hipMemcpyHostToDevice);

    /*
        Shared memory space = blockSize * sizeof(float)
        Copying the entire array of size n to shared memory is insane
    */
    gaussian_kde_kernel<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(n, h, d_x, d_y);

    // Load result back to y
    hipMemcpy(y.data(), d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_x);
    hipFree(d_y);
}